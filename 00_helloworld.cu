
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    printf("Hello from GPU!%d\n", idx);
}

int main()
{
    hello_from_gpu<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}